#include "hip/hip_runtime.h"
/* 
   Original Author of gol_textual.c:
	Christopher Mitchell <chrism@lclark.edu>
   CUDAfied version by Michael Barger <mbarger@pdx.edu>
    for Homework 2 for CS510[GPU] (Prof Karavanic)
 */	

#include "gol_common2.h"


// The two boards -- host only needs one
unsigned char h_current[WIDTH * HEIGHT];
unsigned char *d_current;
size_t current_pitch;
unsigned char *d_next;
size_t next_pitch;

// set the grid/block dimensions for kernel execution
const dim3 gridDim(16, 16, 1);
const dim3 blocksDim(8+2, 8+2, 1); // 256 threads per block
const size_t sharedmem_size = (8+2) * (8+2);

extern __shared__ unsigned char local_current [];

__global__ void step	(const unsigned char *current,	// previous 2D field
						 size_t c_pitch,		// pitch of "current"
						 unsigned char *next,				// result 2D field
						 size_t n_pitch) {		// pitch of "next"
    // register constants for efficiency & readability
    const int local_x = threadIdx.x;
    const int local_y = threadIdx.y;
    const int block_w = blockDim.x;
    const int block_h = blockDim.y;
    const int abs_x = blockIdx.x * (block_w - 2) + local_x - 1;
    const int abs_y = blockIdx.y * (block_h - 2) + local_y - 1;
    const int maxx = block_w - 1;
    const int maxy = block_h - 1;
    const int local_pitch = block_w;
    const int offsets[8][2] = {{-1, 1},{0, 1},{1, 1},
                           {-1, 0},       {1, 0},
                           {-1,-1},{0,-1},{1,-1}};
    
    // normalize 2d array pitches for use with array notation
    c_pitch /= sizeof(int);
    n_pitch /= sizeof(int);
    
    int num_neighbors = 0;	// to numerate the number of neighbors
    int nx, ny;				// to improve readability for temp array index calculations
    
    // populate the shared memory
    nx = (abs_x + WIDTH) % WIDTH;
    ny = (abs_y + HEIGHT) % HEIGHT;
    local_current[local_y * local_pitch + local_x] = current[ny * c_pitch + nx];
    __syncthreads();

	// if a shared-memory builder thread, don't do anything more
    if (local_x > 0 && local_x <= maxx && local_y > 0 && local_y <= maxy) {
        // count this cell's alive neighbors
	    num_neighbors = 0;
	    for (int i = 0; i < 8; i++) {
	        nx = local_x + offsets[i][0];
	        ny = local_y + offsets[i][1];
	        num_neighbors += local_current[ny * local_pitch + nx]==1;
	    }
	
	    // apply the Game of Life rules to this cell
	    next[abs_y * n_pitch + abs_x] = ((local_current[local_y * local_pitch + local_x] && num_neighbors==2) || num_neighbors==3);
	}
}


void loop_func() {
    step<<<gridDim, blocksDim, sharedmem_size>>>(d_current, current_pitch, d_next, next_pitch);
    cudaCheckError("kernel execution");

    hipMemcpy2D(h_current, row_bytes, d_next, next_pitch, row_bytes, HEIGHT, hipMemcpyDeviceToHost);
    cudaCheckError("Device->Host memcpy");

    hipMemcpy2D(d_current, current_pitch, d_next, next_pitch, row_bytes, HEIGHT, hipMemcpyDeviceToDevice);
    cudaCheckError("Device->Device memcpy");
}


int main(void) {
	// allocate the device-side field arrays
	hipMallocPitch((void **)&d_current, &current_pitch, row_bytes, HEIGHT);
	hipMallocPitch((void **)&d_next, &next_pitch, row_bytes, HEIGHT);
	cudaCheckError("device memory allocation");

    // Initialize the host-side "current".
    fill_board(h_current);
    
    // copy host memory to device
    hipMemcpy2D(d_current, current_pitch, h_current, row_bytes, row_bytes, HEIGHT, hipMemcpyHostToDevice);
    cudaCheckError("init array host->device copy");
    
    // run the simulation!
    animate(loop_func, h_current);

	// free device memory
	hipFree(d_current);
	hipFree(d_next);
    return 0;
}
