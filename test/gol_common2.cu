#include "gol_common2.h"

const int field_dim = HEIGHT * WIDTH;
const size_t field_size = field_dim;
const size_t row_bytes = WIDTH;


void fill_board(unsigned char *board) {
    for (int i=0; i<field_dim; i++)
        board[i] = rand() % 2;
}

void print_board(const unsigned char *board) {
    int x, y;
    for (y=0; y<P_HEIGHT; y++) {
        for (x=0; x<P_WIDTH; x++) {
            char c = board[y * WIDTH + x] ? '#':' ';
            //printf("%c", c);
        }
        //printf("\n");
    }
    //printf("-----\n");
}

/* error check routine */
void cudaCheckError (const char *msg) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


void animate(void (*func)(void), const unsigned char *board) {
	int i;
    /*struct timespec delay = {0, 300000000}; // 0.300 seconds*/
    /*struct timespec remaining;*/
/*
    while (1)
    	print_board(board);
    	(*func)();
        nanosleep(&delay, &remaining);
    }
*/   
    for (i = 1; i < 1024; i++) {
    	print_board(board);
    	(*func)();
    }
	printf( "hi!\n" );
}